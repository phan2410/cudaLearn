
#include <hip/hip_runtime.h>
#include <iostream>

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int main(void) {
	hipDeviceProp_t	prop; int dev;

	HANDLE_ERROR(hipGetDevice(&dev));
	printf("ID of current CUDA device:	%d\n", dev);


	memset(&prop, 0, sizeof(hipDeviceProp_t)); 
	prop.major = 6;
	prop.minor = 1;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	printf("ID of CUDA device closest to revision 6.1: %d\n", dev); HANDLE_ERROR(hipSetDevice(dev));

	system("pause");
	return 0;
}
