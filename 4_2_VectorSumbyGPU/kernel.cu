#include "hip/hip_runtime.h"
#include <iostream>
#include "../anLogger/src/anlogger.h"

#define N 79872

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void add(int *a, int *b, int *c) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;	// handle the data at this index
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}


int main(void) {
	double start, end;
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int))); 
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int))); 
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill the arrays 'a' and 'b' on the CPU
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	
	
	// copy the arrays 'a' and 'b' to the GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int),hipMemcpyHostToDevice)); 
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int),hipMemcpyHostToDevice)); 
	dim3 blocksPerGrid(N/32,1,1);
	dim3 threadsPerBlock(32,1,1);
	start = __anElapsedTimeNSEC__;
	add << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_c);
	//add << <N, 1 >> > (dev_a, dev_b, dev_c);
	
	// copy the array 'c' back from the GPU to the CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost));
	end = __anElapsedTimeNSEC__;
	// display the results
	//for (int i = 0; i < N; i++) {
	//	printf("%d + %d = %d\n", a[i], b[i], c[i]);
	//}


	// free the memory allocated on the GPU
	hipFree(dev_a); 
	hipFree(dev_b); 
	hipFree(dev_c);

	printf("Vector sumed by GPU consumes %lf ns.\n",(end - start));
	system("pause");
	return 0;
}