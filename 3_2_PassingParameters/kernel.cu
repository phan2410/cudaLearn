#include "hip/hip_runtime.h"
#include <iostream>



static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}


int main(void) {
	int c;
	int *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int))); 
	add << <1, 1 >> >(2, 7, dev_c);

	HANDLE_ERROR(hipMemcpy(&c,dev_c, sizeof(int),hipMemcpyDeviceToHost)); 
	printf("2 + 7 = %d\n", c);

	hipFree(dev_c);		

	system("pause");
	return 0;
}
